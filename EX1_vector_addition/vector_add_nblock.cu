#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N 268435456

__global__ void vector_add(float *out, float *a, float *b, int n){
     int index = blockIdx.x *blockDim.x + threadIdx.x; 
     if (index<n){
     out[index] = a[index] + b[index];}
}

int main(){
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    a   = (float*)malloc(sizeof(float) * N); 
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; 
        b[i] = 2.0f;
    }
    
    // Allocate device memore for a
    hipMalloc((void**)&d_a,sizeof(float)*N);
    hipMalloc((void**)&d_b,sizeof(float)*N);
    hipMalloc((void**)&d_out,sizeof(float)*N);
 
    // Transfer data from host to device memory
    hipMemcpy(d_a,a, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(d_b,b, sizeof(float)*N, hipMemcpyHostToDevice);

    // Main function
    int block_size = 256;
    int grid_size  = (N+block_size)/block_size; 
    vector_add<<<grid_size,block_size>>>(d_out, d_a, d_b, N);

    hipMemcpy(out, d_out, sizeof(float)*N, hipMemcpyDeviceToHost);

    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Deallocate host memory
    free(a); 
    free(b); 
    free(out);

}
